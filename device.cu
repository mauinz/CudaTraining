#include "hip/hip_runtime.h"

#include <stdio.h>

int main(){
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  printf("\nNumber of GPU devices: %i\n", deviceCount);
  int driverVersion;
  int runtimeVersion;
  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);
  hipDeviceProp_t deviceProperties;
  printf("CUDA Driver Version / Runtime Version: %d.%d / %d.%d\n\n\n",
  driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000,
  (runtimeVersion%100)/10);

  for(int i=0; i<deviceCount; i++)
  {
    hipGetDeviceProperties(&deviceProperties, i);
    printf("Name: %s\n", deviceProperties.name);
  }

  return 0;
}
