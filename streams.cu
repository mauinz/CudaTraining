#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

const N = 4096000
__global__ void mulKernel(int *a, int *c)
{
  int tdx = blockIdx.x * blockDim.x + threadIdx.x;
  if(tdx < N)
  {
    c[tdx] = a[tdx]*2;
  }
}

int main()
{

  int *a_h[2], *c_h[2];
  //device memory pointers
  int *a_d[2];
  int *c_d[2];
  hipStream_t stream[2];
  for (int i = 0; i < 2; ++i)
  {
    hipStreamCreate(&stream[i]); //stream creation
    //pinned memory allocation
    hipHostMalloc((void**)&a_h[i], (N/2)*sizeof(int));
    hipHostMalloc((void**)&c_h[i], (N/2)*sizeof(int));
    //allocate device memory
    hipMalloc((void**)&a_d[i], (N/2)*sizeof(int));
    hipMalloc((void**)&b_d[i], (N/2)*sizeof(int));

    hipMalloc((void**)&c_d[i], (N/2)*sizeof(int));
  }
  //load arrays with some numbers
  for(int i=0; i<2; i++)
  {
    for(int ii=0; ii<N/2; ii++)
    {
      a_h[i][ii] = i*N/2+ii;
    }
  }
  //CUDA events to measure time
  hipEvent_t start;
  hipEvent_t stop;
  float elapsedTime;
  //start timer
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  dim3 grid(4,1,1);
  dim3 block(1024,1,1);
  //stream 2
  hipMemcpyAsync(a_d[0], a_h[0], (N/2)*sizeof(int), hipMemcpyHostToDevice, stream[0]);
  vectorAddKernel <<< grid, block, 0, stream[0]>>>(a_d[0], c_d[0]);
  hipMemcpyAsync(c_h[0], c_d[0], (N/2)*sizeof(int), hipMemcpyDeviceToHost, stream[0]);
  //stream 1
  hipMemcpyAsync(a_d[1], a_h[1], (N/2)*sizeof(int), hipMemcpyHostToDevice, stream[1]);
  vectorAddKernel <<<grid, block, 0, stream[1]>>>(a_d[1], c_d[1]);
  hipMemcpyAsync(c_h[1], c_d[1], (N/2)*sizeof(int), hipMemcpyDeviceToHost, stream[1]);
  
  //stop timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  //print out execution time
  printf("Time to calculate results: %f ms.\n", elapsedTime);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  for (int i = 0; i < 2; ++i)
  {
    hipStreamDestroy(stream[i]);
    //clean up
    hipHostFree(a_h[i]);
    hipHostFree(c_h[i]);
  }
  hipDeviceReset();
  return 0;
}
